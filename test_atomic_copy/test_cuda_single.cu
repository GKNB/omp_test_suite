#include<omp.h>
#include<cstdlib>
#include<iostream>
#include<cstdlib>
#include<cassert>

#include<hip/hip_runtime.h>

template<typename T>
__global__ void test_atomic_add(T* res, T* data, int size)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < size)
  {
    atomicAdd(res, data[i]);
  }
}

int main()
{
  constexpr int N = 1024 * 1024 * 32;
  constexpr int nlp = 10;

  using dataType = int;
  dataType delta = (dataType)2e-2;
  std::cout << "We are testing atomicAdd with type " << typeid(delta).name() << std::endl;
  srand(1234);

  dataType* data = (dataType*)malloc(sizeof(dataType) * N);
  for(int i=0; i<N; i++)
  {
    data[i] = rand() / (dataType)RAND_MAX;
  }

  dataType* res   = (dataType*)malloc(sizeof(dataType));
  dataType* res_h = (dataType*)malloc(sizeof(dataType));
  *res = *res_h = 0.0;

  dataType* res_d;
  hipMalloc((void**)&res_d, sizeof(dataType));
  hipMemcpy(res_d, res_h, sizeof(dataType), hipMemcpyHostToDevice);

  dataType* data_d;
  hipMalloc((void**)&data_d, sizeof(dataType) * N);
  hipMemcpy(data_d, data, sizeof(dataType) * N, hipMemcpyHostToDevice);

  //=====================================Serial running==================================
  double tc = 0.0;
  for(int lp = 0; lp < nlp; lp++)
  {
    tc -= omp_get_wtime();
    for(int i=0; i<N; i++)
      res[0] += data[i];
    tc += omp_get_wtime();
  }
  std::cout << "Time for serial with N = " << N << " is " << tc << std::endl;

  //=====================================CUDA running==================================
  tc = 0.0;
  tc -= omp_get_wtime();
  for(int lp = 0; lp < nlp; lp++)
  {
    test_atomic_add<<<N/512,512>>>(res_d, data_d, N);
  }
  hipDeviceSynchronize();
  tc += omp_get_wtime();

  hipMemcpy(res_h, res_d, sizeof(dataType), hipMemcpyDeviceToHost);

  bool do_abort = false;
  
  if(abs(*res_h - *res) > delta)
  {
    do_abort = true;
  }
  
  if(do_abort)
    assert(0 && "Error! Atomic update for CUDA fail\n");

  std::cout << "Time for CUDA with N = " << N << " is " << tc << std::endl;

  return 0;
}
