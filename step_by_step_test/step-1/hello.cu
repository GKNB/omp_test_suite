
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <cstdlib>

__global__ void hello()
{
  printf("Hello from block: %u, thread: %u\n", blockIdx.x, threadIdx.x);
}

//Add this function since printf on cori gpu does not seem working
__global__ void easy(float *da, int n)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if(idx < n)
    da[idx] = da[idx] * 3.0;
}

int main(){

  int n = 64;
  hello<<<2,2>>>();

  float *ha;
  ha = (float*)malloc(sizeof(float) * n);
  for(int i=0; i<n; i++)
    ha[i] = float(i);

  float *da;
  hipMalloc(&da, sizeof(float) * n);
  hipMemcpy(da, ha, sizeof(float) * n, hipMemcpyHostToDevice);
  easy<<<2,32>>>(da, n);

  hipMemcpy(ha, da, sizeof(float) * n, hipMemcpyDeviceToHost);
  for(int i=0; i<n; i++)
    std::cout << ha[i] << std::endl;

  return 0;
}
