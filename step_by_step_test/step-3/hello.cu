
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipblas.h>
#include <iostream>
#include <cstdlib>

__global__ void hello()
{
  printf("Hello from block: %u, thread: %u\n", blockIdx.x, threadIdx.x);
}

//__global__ void myAxpy(int n, float a, float *x, float *y)
//{
//  int idx = threadIdx.x + blockIdx.x * blockDim.x;
//  if(idx < n)
//    y[idx] = y[idx] + a * x[idx];
//}

int main()
{
  hello<<<2,2>>>();

  float *x = (float*)malloc(sizeof(float) * 1000);
  float *y = (float*)malloc(sizeof(float) * 1000);
  for(int i=0; i<1000; i++)
  {
    x[i] = float(i);
    y[i] = 3.0f * float(i);
  }

  float *dx;
  float *dy;
  hipMalloc(&dx, sizeof(float) * 1000);
  hipMalloc(&dy, sizeof(float) * 1000);

  hipMemcpy(dx, x, sizeof(float) * 1000, hipMemcpyHostToDevice);
  hipMemcpy(dy, y, sizeof(float) * 1000, hipMemcpyHostToDevice);

  float a = 1.5f;

  hipblasStatus_t stat;
  hipblasHandle_t handle;
  stat = hipblasCreate(&handle);

  stat = hipblasSaxpy(handle, 1000, &a, dx, 1, dy, 1);

  hipMemcpy(y, dy, sizeof(float) * 1000, hipMemcpyDeviceToHost);

  std::cout << "y[0] = " << y[0] << "\ny[45] = " << y[45] << "\ny[999] = " << y[999] << std::endl;
  
  return 0;
}
