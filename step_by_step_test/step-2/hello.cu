
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib>
#include <iostream>

__global__ void hello()
{
  printf("Hello from block: %u, thread: %u\n", blockIdx.x, threadIdx.x);
}

__global__ void myAxpy(int n, float a, float *x, float *y)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if(idx < n)
    y[idx] = y[idx] + a * x[idx];
}

int main()
{
  hello<<<2,2>>>();

  float *x = (float*)malloc(sizeof(float) * 1000);
  float *y = (float*)malloc(sizeof(float) * 1000);
  for(int i=0; i<1000; i++)
  {
    x[i] = float(i);
    y[i] = 3.0f * float(i);
  }

  float *dx;
  float *dy;
  hipMalloc(&dx, sizeof(float) * 1000);
  hipMalloc(&dy, sizeof(float) * 1000);

  hipMemcpy(dx, x, sizeof(float) * 1000, hipMemcpyHostToDevice);
  hipMemcpy(dy, y, sizeof(float) * 1000, hipMemcpyHostToDevice);

  myAxpy<<<32,256>>>(1000, 1.5, dx, dy);

  hipMemcpy(y, dy, sizeof(float) * 1000, hipMemcpyDeviceToHost);

  std::cout << "y[0] = " << y[0] << "\ny[45] = " << y[45] << "\ny[999] = " << y[999] << std::endl;

  return 0;
}
